#include <chrono>
#include <hip/hip_runtime.h>
#include <iostream>

/// this configuration (2000000ul*32*1024) need about 1 seconds in my GTX 1060 6GB
/// find the result need about 34minutes,😄
const uint64_t chucksize = 2000000ul;
const uint gridsize      = 32;
const uint blocksize     = 1024;

__constant__ __device__ uint64_t prog[16] = {2, 4, 1, 5, 7, 5, 1, 6, 0, 3, 4, 6, 5, 5, 3, 0};

__global__ void check_num_kernel(uint64_t start, uint64_t* d_result, bool* d_found, uint64_t* d_found_i) {
    uint64_t tid     = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t start_i = start + tid * chucksize;

    if (*d_found && start_i > *d_found_i)
        return;

    for (uint64_t i = start_i; i < start_i + chucksize; ++i) {
        uint64_t r0 = i;
        uint64_t r1 = 0;
        uint64_t r2 = 0;
        int step    = 0;

        while (step < 16) {
            r1 = r0 % 8;
            r1 ^= 5;
            r2 = r0 >> r1;
            r1 ^= 6;
            r0 >>= 3;
            r1 ^= r2;
            if ((r1 & 7) != prog[step]) {
                break;
            }
            if (r0 == 0)
                break;

            ++step;
        }

        if (step == 15) {
            atomicMin((unsigned long long*)d_result, i);
            *d_found   = true;
            *d_found_i = i;
        }
    }
}

int main() {
    bool* h_found = new bool(false);
    bool* d_found;

    uint64_t* h_found_i = new uint64_t(UINT64_MAX);
    uint64_t* d_found_i;

    uint64_t* h_result = new uint64_t(UINT64_MAX);
    uint64_t* d_result;

    hipMalloc(&d_result, sizeof(uint64_t));
    hipMalloc(&d_found, sizeof(bool));
    hipMalloc(&d_found_i, sizeof(uint64_t));
    hipMemcpy(d_result, h_result, sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(d_found, h_found, sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(d_found_i, h_found_i, sizeof(uint64_t), hipMemcpyHostToDevice);

    std::cout << "batch size: " << gridsize * blocksize * chucksize << std::endl;
    auto starttime = std::chrono::high_resolution_clock::now();

    for (size_t i = 0; i < UINT64_MAX; i += gridsize * blocksize * chucksize) {
        if (*h_found && i > *h_found_i) {
            break;
        }
#ifndef NDEBUG
        auto cuda_start_time = std::chrono::high_resolution_clock::now();
#endif
        check_num_kernel<<<gridsize, blocksize>>>(i, d_result, d_found, d_found_i);
        hipDeviceSynchronize();

        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
            std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;

        hipMemcpy(h_result, d_result, sizeof(uint64_t), hipMemcpyDeviceToHost);
        hipMemcpy(h_found, d_found, sizeof(bool), hipMemcpyDeviceToHost);
        hipMemcpy(h_found_i, d_found_i, sizeof(uint64_t), hipMemcpyDeviceToHost);

#ifndef NDEBUG
        auto cuda_end_time = std::chrono::high_resolution_clock::now();
        auto durationms    = std::chrono::duration_cast<std::chrono::milliseconds>(cuda_end_time - cuda_start_time);
        std::cout << durationms.count() << "ms" << " " << i << std::endl;
#endif
    }

    std::cout << *h_result << std::endl;
    auto endtime    = std::chrono::high_resolution_clock::now();
    auto durationms = std::chrono::duration_cast<std::chrono::minutes>(endtime - starttime);
    std::cout << durationms.count() << " minutes" << std::endl;

    delete h_result;
    delete h_found;
    delete h_found_i;
    hipFree(d_result);
    hipFree(h_found);
    hipFree(h_found_i);
    return 0;
}
