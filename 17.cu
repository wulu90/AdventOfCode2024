#include <hip/hip_runtime.h>
#include <iostream>
#include <limits>

__constant__ __device__ int prog[16] = {2, 4, 1, 5, 7, 5, 1, 6, 0, 3, 4, 6, 5, 5, 3, 0};

__global__ void check_num_kernel(unsigned long long n, unsigned long long* result) {
    unsigned long long r0 = n + blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long r1 = 0;
    unsigned long long r2 = 0;
    int step              = 0;

    while (step < 16) {
        r1 = r0 % 8;
        r1 ^= 5;
        r2 = r0 >> r1;
        r1 ^= 6;
        r0 >>= 3;
        r1 ^= r2;
        if (r1 % 8 != prog[step]) {
            break;
        }
        if (r0 == 0)
            break;

        ++step;
    }

    if (step == 15) {
        *result = n + blockIdx.x * blockDim.x + threadIdx.x;
    }
}

int main() {
    // char* h_results = (char*)malloc(1024 * 32);
    unsigned long long* h_result = new unsigned long long(0);
    unsigned long long* d_result;
    hipMalloc(&d_result, sizeof(unsigned long long));
    // 35184372088832
    for (size_t i = 106904920099226; i < std::numeric_limits<size_t>::max(); i += 1024ul * 2147483647) {
        // for (size_t i = 0; i < 1024 * 32 * 10; i += 1024 * 32) {
        hipMemset(d_result, 0, sizeof(unsigned long long));
        check_num_kernel<<<2147483647, 1024>>>(i, d_result);
        hipDeviceSynchronize();
        hipMemcpy(h_result, d_result, sizeof(unsigned long long), hipMemcpyDeviceToHost);
        if (*h_result != 0) {
            std::cout << *h_result << std::endl;
            break;
        }
    }
    delete h_result;
    hipFree(d_result);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
    // cudaProfilerStop();
}
